#include "hip/hip_runtime.h"
#include <wb.h>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__device__ int binarySearch(const int value, const int *A, const int N)
{
	// TODO: Implement a binary search that returns
	// the index where all values in A are less than
	// the given value.

	return 0;
}

__device__ int linearSearch(const int value, const int *A, const int N)
{
	// TODO: Implement a sequential search that returns
	// the index where all values in A are less than
	// the given value.
	int i;
	
	for (i = 0; i < N; i++)
	{
		if (A[i] > value)
			break;
	}

	return i;
}

__global__ void merge(int *C, const int *A, const int *B, const int N)
{
	// TODO: Merge arrays A and B into C. To make it
	// easier you can assume the following:
	// 
	// 1) A and B are both size N
	//
	// 2) C is size 2N
	//
	// 3) Both A and B are sorted arrays
	//
	// The algorithm should work as follows:
	// Given inputs A and B as follows:
	// A = [0 2 4 10]
	// B = [1 5 7 9]

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < N)
	{
		int j = linearSearch(A[i], B, N);

		int k = linearSearch(B[i], A, N);

		C[i+j] = A[i];
		C[i+k] = B[i];

	}
	//
	// Step 1:
	// Find for each element in array A the index i that
	// would A[i] be inserted in array B or in other 
	// words find the smallest j where A[i] < B[j].
	//
	// Step 2:
	// Do the same for B, but this time find the j 
	// where B[i] < A[j].
	//
	// Step 3:
	// Since we know how many elements come before
	// A[i] in array A and we know how many elements 
	// come before A[i] in array B, which is given by
	// are calculation of j. We should know where A[i]
	// is inserted into C, given i and j.
	//
	// This same logic can be used to find where B[i]
	// should be inserted into C. Although you will have
	// to make a minor change to handle duplicates in A 
	// and B. Or in other words if A and B intersect at 
	// all some values in C will be incorrect. This 
	// occurs because A and B will want to put the values 
	// in the same place in C.
	
}

int main(int argc, char **argv) {
	wbArg_t args;
	int N;
	int* A;
	int* B;
	int* C;
	int* deviceA;
	int* deviceB;
	int* deviceC;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	A = (int *)wbImport(wbArg_getInputFile(args, 0), &N, NULL, "Integer");
	B = (int *)wbImport(wbArg_getInputFile(args, 1), &N, NULL, "Integer");
	C = (int *)malloc(2 * N * sizeof(int));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", N);

	int threads = 256;
	int blocks = N / threads + ((N%threads == 0) ? 0 : 1);

	wbTime_start(GPU, "Allocating GPU memory.");
	hipMalloc((void **)&deviceA, N * sizeof(int));
	hipMalloc((void **)&deviceB, N * sizeof(int));
	hipMalloc((void **)&deviceC, 2 * N * sizeof(int));
	wbTime_stop(GPU, "Allocating GPU memory.");


	wbTime_start(GPU, "Copying input memory to the GPU.");
	hipMemcpy(deviceA, A, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceB, B, N * sizeof(int), hipMemcpyHostToDevice);
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// Perform on CUDA.
	const dim3 blockSize(threads, 1, 1);
	const dim3 gridSize(blocks, 1, 1);

	wbTime_start(Compute, "Performing CUDA computation");
	merge << < gridSize, blockSize >> >(deviceC, deviceA, deviceB, N);
	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	hipMemcpy(C, deviceC, 2 * N * sizeof(int), hipMemcpyDeviceToHost);
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);
	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, C, 2*N);

	free(A);
	free(B);
	free(C);

#if LAB_DEBUG
	system("pause");
#endif

	return 0;
}
